#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include "ops.cuh"

namespace black_manbo
{
	__global__ void add_kernel(float* a, float* b, float* c, int N)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if(idx < N)
			c[idx] = a[idx] + b[idx];
	}

	void add_kernel_launcher(float* d_a, float* d_b, float* d_c, int N)
	{
		add_kernel<<<(N + 255) / 256, 256>>>(d_a, d_b, d_c, N);
		hipDeviceSynchronize();
	}
}