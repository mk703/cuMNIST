#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>
#include "ops.cuh"

//row y M, col x N

namespace black_manbo
{
	__global__ void matmul_kernel(float* a, float* b, float* c, int M, int N, int K)
    {
        extern __shared__ float shared_mem[];
		float* shared_a = shared_mem;
		float* shared_b = shared_mem + blockDim.y * blockDim.x;

		int row = blockIdx.y * blockDim.y + threadIdx.y;
		int col = blockIdx.x * blockDim.x + threadIdx.x;

		float sum = 0.0f;

		for (int i = 0; i < (K + blockDim.x - 1) / blockDim.x; i++)
		{
			if (row < M && i * blockDim.x + threadIdx.x < K)
				shared_a[threadIdx.y * blockDim.x + threadIdx.x] = a[row * K + i * blockDim.x + threadIdx.x];
			else
				shared_a[threadIdx.y * blockDim.x + threadIdx.x] = 0.0f;

			if (col < N && i * blockDim.y + threadIdx.y < K)
				shared_b[threadIdx.y * blockDim.x + threadIdx.x] = b[(i * blockDim.y + threadIdx.y) * N + col];
			else
				shared_b[threadIdx.y * blockDim.x + threadIdx.x] = 0.0f;

			__syncthreads();

			for (int j = 0; j < blockDim.x; j++)
				sum += shared_a[threadIdx.y * blockDim.x + j] * shared_b[j * blockDim.x + threadIdx.x];

			__syncthreads();
		}

		if (row < M && col < N)
			c[row * N + col] = sum;
    }

    void matmul_kernel_launcher(float* d_a, float* d_b, float* d_c, int M, int N, int K)
    {
        dim3 blockDim(16, 16);  
        dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);
        size_t shared_mem_size = (blockDim.y * blockDim.x * 2) * sizeof(float);
        matmul_kernel<<<gridDim, blockDim, shared_mem_size>>>(d_a, d_b, d_c, M, N, K);
        hipDeviceSynchronize();
	}
}