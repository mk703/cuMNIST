#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include "ops.cuh"
#include "optional_ops/SoftMax.cuh"

namespace manbo
{
	__global__ void SoftMax::softmax_kernel(float* __restrict__ a, float* __restrict__ b, int N)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if(idx >= N) return;
		float max = a[idx];
		for(int i = 0; i < N; i++)
			if(a[i] > max)
				max = a[i];
		float sum = 0;
		for(int i = 0; i < N; i++)
		{
			b[i] = expf(a[i] - max);
			sum += b[i];
		}
		for(int i = 0; i < N; i++)
			b[i] /= sum;
	}

	void SoftMax::softmax_kernel_launcher(float* d_a, float* d_b, int N, int M)
	{
		for(int i = 0;i < N / M; i ++)
			softmax_kernel<<<(M + 255) / 256, 256>>>(d_a + i * M, d_b + i * M, M);
		hipDeviceSynchronize();
	}
}