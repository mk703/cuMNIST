#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include "ops.cuh"
#include "optional_ops/SoftMax.cuh"

namespace manbo
{
	__global__ void SoftMax::softmax_kernel(float* __restrict__ a, float* __restrict__ b, float* max, int N)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if(idx >= N) return;
		//用原子操作更新max
		atomicMax(max, a[idx]);
		__syncthreads();

		for(int i = 0; i < N; i++)
			if(a[i] > &max)
				max = a[i];
		float sum = 0;
		for(int i = 0; i < N; i++)
		{
			b[i] = expf(a[i] - max);
			sum += b[i];
		}
		for(int i = 0; i < N; i++)
			b[i] /= sum;
	}

	void SoftMax::softmax_kernel_launcher(float* d_a, float* d_b, int N, int M)
	{
		for(int i = 0;i < N / M; i ++)
			softmax_kernel<<<(M + 255) / 256, 256>>>(d_a + i * M, d_b + i * M, M);
		hipDeviceSynchronize();
	}
}