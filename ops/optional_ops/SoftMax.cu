#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include "ops.cuh"

namespace black_manbo
{
	__global__ void softmax_kernel(float* __restrict__ a, float* __restrict__ b, int N)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if(idx >= N) return;
		float max = a[idx];
		for(int i = 0; i < N; i++)
			if(a[i] > max)
				max = a[i];
		float sum = 0;
		for(int i = 0; i < N; i++)
		{
			b[i] = expf(a[i] - max);
			sum += b[i];
		}
		for(int i = 0; i < N; i++)
			b[i] /= sum;
	}

	void softmax_kernel_launcher(float* d_a, float* d_b, int N, int M)
	{
		for(int i = 0;i < N / M; i ++)
			softmax_kernel<<<(M + 255) / 256, 256>>>(d_a + i * M, d_b + i * M, M);
		hipDeviceSynchronize();
	}
}