#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include "ops.cuh"
#include "optional_ops/add.cuh"

namespace manbo
{
	__global__ void Add::add_kernel(const float* __restrict__ a, const float* __restrict__ b, float* __restrict__ c, int N)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if(idx < N)
			c[idx] = a[idx] + b[idx];
	}

	void Add::add_kernel_launcher(float* d_a, float* d_b, float* d_c, int N)
	{
		add_kernel<<<(N + 255) / 256, 256>>>(d_a, d_b, d_c, N);
		hipDeviceSynchronize();//等待所有线程结束
	}
}