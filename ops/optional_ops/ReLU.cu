#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include "ops.cuh"
#include "optional_ops/ReLU.cuh"

namespace manbo
{
	__global__ void ReLU::relu_kernel(const float* __restrict__ a, float* __restrict__ b, int N)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if(idx < N)
			b[idx] = fmaxf(0, a[idx]);
	}


	void ReLU::relu_kernel_launcher(float* d_a, float* d_b, int N)
	{
		relu_kernel<<<(N + 255) / 256, 256>>>(d_a, d_b, N);
		hipDeviceSynchronize();
	}
}