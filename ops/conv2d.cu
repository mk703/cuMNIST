#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include "ops.cuh"

namespace black_manbo
{
	__global__ void conv2d_kernel(const float* d_input, float* d_output, const float* d_kernel, int width, int height, int kernelSize)
	{
		int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;
		int half = kernelSize / 2;

		extern __shared__ float shared_memory[];
		float* shared_kernel = shared_memory;
		float* shared_input = &shared_memory[kernelSize * kernelSize];


		if(threadIdx.x < kernelSize && threadIdx.y < kernelSize)
			shared_kernel[threadIdx.y * kernelSize + threadIdx.x] = d_kernel[threadIdx.y * kernelSize + threadIdx.x];
		
		int shared_width = blockDim.x + kernelSize - 1;
		int shared_x = threadIdx.x + half;
		int shared_y = threadIdx.y + half;

		if (x < width && y < height) 
			shared_input[shared_y * shared_width + shared_x] = d_input[y * width + x];
		else
			shared_input[shared_y * shared_width + shared_x] = 0.0f;
		
		if (threadIdx.x < half)
		{
			int left_x = max(x - half, 0);
			shared_input[shared_y * shared_width + threadIdx.x] = d_input[y * width + left_x];
			int right_x = min(x + blockDim.x, width - 1);
			shared_input[shared_y * shared_width + shared_x + blockDim.x] = d_input[y * width + right_x];
		}
		if (threadIdx.y < half)
		{
			int top_y = max(y - half, 0);
			shared_input[threadIdx.y * shared_width + shared_x] = d_input[top_y * width + x];
			int bottom_y = min(y + blockDim.y, height - 1);
			shared_input[(shared_y + blockDim.y) * shared_width + shared_x] = d_input[bottom_y * width + x];
		}
		if(threadIdx.x < half && threadIdx.y < half)
		{
			shared_input[threadIdx.y * shared_width + threadIdx.x] = d_input[max(y - half, 0) * width + max(x - half, 0)];
			shared_input[threadIdx.y * shared_width + shared_x + blockDim.x] = d_input[max(y - half, 0) * width + min(x + blockDim.x, width - 1)];
			shared_input[(shared_y + blockDim.y) * shared_width + threadIdx.x] = d_input[min(y + blockDim.y, height - 1) * width + max(x - half, 0)];
			shared_input[(shared_y + blockDim.y) * shared_width + shared_x + blockDim.x] = d_input[min(y + blockDim.y, height - 1) * width + min(x + blockDim.x, width - 1)];
		}
		__syncthreads();

		if(x >= width || y >= height)
			return;
		float sum = 0.0f;
		for(int ky = -half; ky <= half; ky++)
			for(int kx = -half; kx <= half; kx++)
			{
				int shared_ix = shared_x + kx;
					int shared_iy = shared_y + ky;
					sum += shared_input[shared_iy * shared_width + shared_ix] * shared_kernel[(ky + half) * kernelSize + (kx + half)];
			}
		d_output[y * width + x] = sum;
	}

	void conv2d_kernel_launcher(const float* d_input, float* d_output, const float* d_kernel, int width, int height, int kernelSize)
	{
		dim3 block(16, 16);
		dim3 grid((width + 15) / 16, (height + 15) / 16);
		size_t shared_memory_size = (kernelSize * kernelSize + (width + kernelSize - 1) * (height + kernelSize - 1)) * sizeof(float);
		conv2d_kernel<<<grid, block, shared_memory_size>>>(
			d_input, d_output, d_kernel,
			width, height, kernelSize);
		hipDeviceSynchronize();
	}
}