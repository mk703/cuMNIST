#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include "ops.cuh"

namespace black_manbo
{
	__global__ void relu_kernel(float* a, float* b, int N)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if(idx < N)
			b[idx] = fmaxf(0, a[idx]);
	}


	void relu_kernel_launcher(float* d_a, float* d_b, int N)
	{
		relu_kernel<<<(N + 255) / 256, 256>>>(d_a, d_b, N);
		hipDeviceSynchronize();
	}
}