#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include "ops.cuh"

namespace black_manbo
{
	__global__ void softmax_kernel(float* a, float* b, int N)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if(idx >= N) return;
		float max = a[idx];
		for(int i = 0; i < N; i++)
			if(a[i] > max)
				max = a[i];
		float sum = 0;
		for(int i = 0; i < N; i++)
		{
			b[i] = expf(a[i] - max);
			sum += b[i];
		}
		for(int i = 0; i < N; i++)
			b[i] /= sum;
	}

	void softmax_kernel_launcher(float* d_a, float* d_b, int N)
	{
		softmax_kernel<<<(N + 255) / 256, 256>>>(d_a, d_b, N);
		hipDeviceSynchronize();
	}
}