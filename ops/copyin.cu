#include <hip/hip_runtime.h>
#include <math.h>
#include "ops.cuh"
#ifndef COPYIN
#define COPYIN

#include <vector>

namespace manbo
{
	class CopyH2D : public ops
	{
		public:
			CopyH2D(float * in, int in_size, int N)//需要保证in_size是N的整数倍
			{
				this->in = in;
				this->in_size = in_size;
				this->N = N;
			}
			bool execable()
			{
				if(times * N >= in_size || out[0]->data == NULL)
					return false;
				return true;
			}
			void exec()
			{
				hipMemcpy(out[0]->data, in + times * N, N * sizeof(float), hipMemcpyHostToDevice);
				times++;
			}
			float *in;
		private:
			int N, in_size;
			int times;
	};
}

#endif