#include <hip/hip_runtime.h>
#include <math.h>
#include "ops.cuh"
#ifndef COPYIN
#define COPYIN

#include <vector>

namespace manbo
{
	class copyin : public ops
	{
		public:
			void init(float * in, int in_size, int N)
			{
				this->in = in;
				this->in_size = in_size;
				this->N = N;
			}
			void exec()
			{
				hipMemcpyAsync(out[0]->data, in + times * N, N * sizeof(float), hipMemcpyHostToDevice, stream);
				times++;
			}
			float *in;
		private:
			int N, in_size;
			int times;
	};
}

#endif