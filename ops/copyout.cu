#include <hip/hip_runtime.h>
#include <math.h>
#include "ops.cuh"
#ifndef COPYOUT
#define COPYOUT

#include <vector>

namespace manbo
{
	class CopyD2H : public ops
	{
		public:
			CopyD2H(float * out, int out_size, int N)//需要保证out_size是N的整数倍
			{
				this->out = out;
				this->out_size = out_size;
				this->N = N;
			}
			bool execable()
			{
				if(times * N >= out_size || in[0]->data == NULL)
					return false;
				return true;
			}
			void exec()
			{
				hipMemcpy(out + times * N, in[0]->data, N * sizeof(float), hipMemcpyDeviceToHost);
				times++;
			}
			float *out;//在主机上的输出
		private:
			int N, out_size;
			int times;
	};
}

#endif