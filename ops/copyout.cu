#include <hip/hip_runtime.h>
#include <math.h>
#include "ops.cuh"
#ifndef COPYOUT
#define COPYOUT

#include <vector>

namespace manbo
{
	class copyout : public ops
	{
		public:
			void init(float * out, int N)
			{
				this->in = in;
				this->N = N;
			}
			void exec()
			{
				hipMemcpyAsync(out[0]->data, in + times * N, N * sizeof(float), hipMemcpyHostToDevice, stream);
				times++;
			}
			float *in;
		private:
			int N;
			int times;
	};
}

#endif